
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecMatMul(float *A,float *B, float *C,int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<N){
        float sum = 0.0f;
        for (int j = 0; j<N; j++){
            sum += A[i*N + j] * B[j];
        }
        C[i] = sum;
    }
}

int main(){
    int N = 10;
    float *A,*B,*C;
    A = (float *)malloc(N*N*sizeof(float));
    B = (float *)malloc(N*N*sizeof(float));
    C = (float *)malloc(N*N*sizeof(float));

    for (int i = 0; i< N; i++){
        for (int j = 0; j<N;j++){
            A[i*N + j] = 1.0f;
        }
        B[i] = 2.0f;
        C[i] = 0.0f;
        }
    
    float *A_d, *B_d, *C_d;
    hipMalloc(&A_d,N*N*sizeof(float));
    hipMalloc(&B_d,N*sizeof(float));
    hipMalloc(&C_d,N*sizeof(float));
    hipMemcpy(A_d,A,N*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(B_d,B,N*sizeof(float),hipMemcpyHostToDevice);
    int BlockSize = 1024;
    int gridsize = (N+BlockSize-1)/ BlockSize;
    vecMatMul<<<gridsize,BlockSize>>>(A_d,B_d,C_d,N);
    hipDeviceSynchronize();
    hipMemcpy(C,C_d,N*sizeof(float),hipMemcpyDeviceToHost);
    printf("A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {

            printf("%.2f ", A[i * N + j]); 
        }
        printf("\n"); 
    }
    printf("\n");
     printf("B:\n");
    for (int i = 0; i < N; i++) {


            printf("%.2f ", B[i ]); 

    }
    printf("\n");
    printf("C:\n");
    for (int i = 0; i < N; i++) {


            printf("%.2f ",C[i]); 

    }
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}