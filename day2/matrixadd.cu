
#include <hip/hip_runtime.h>
#include<iostream>
__global__ void Matrixadd_A(const float* A, const float*B, float* C, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i<N){
        for (int j = 0; j < N; j++){
            int pos = i*N + j;
            C[pos] = A[pos] + B[pos];
        }
    }
}

__global__ void Matrixadd_B(const float* A,const float *B,float *C, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if ((i>=N) && (j>=N)){return ;}

    int pos = i*N + j;
    C[pos] = A[pos] + B[pos];
}

__global__ void Matrixadd_C(const float* A,const float *B,float *C, int N){
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (j<N){
        for (int i = 0; i < N; i++){
            int pos = i*N + j;
            C[pos] = A[pos] + B[pos];
        }
    }
}

void printmatrix(const char* name, const float* matrix, int N){
    printf("%s:\n", name);
    for (int i =0;i<N; i++){
        for (int j = 0;j<N;j++){
            printf("%.2f", matrix[i*N + j]);
        }
        printf("\n");
    }
}

int main() {
    const int N = 5;
    float *A, *B, *C;
    A = (float*)malloc(N*N* sizeof(float));
    B = (float*)malloc(N*N* sizeof(float));
    C = (float*)malloc(N*N* sizeof(float));
    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            int pos = i*N + j;
            A[pos] = 1.0f;
            B[pos] = 2.0f;
            C[pos] = 0.0f;
        }
    }

    float *A_d, *B_d,*C_d;
    hipMalloc((void **)&A_d,N*N*sizeof(float));
    hipMalloc((void **)&B_d,N*N*sizeof(float));
    hipMalloc((void **)&C_d,N*N*sizeof(float));

    hipMemcpy(A_d,A,N*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(B_d,B,N*N*sizeof(float),hipMemcpyHostToDevice);

    dim3 dimBlock(32,16);
    dim3 dimGrid(ceil(N/32.0f),ceil(N/16.0f));
    Matrixadd_A<<<dimGrid,dimBlock>>>(A_d,B_d,C_d,N);
    hipDeviceSynchronize();

    hipMemcpy(C, C_d, N*N*sizeof(float), hipMemcpyDeviceToHost);
    printmatrix("C",C,N);
    printmatrix("A",A,N);
    printmatrix("B",B,N);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    free(A);
    free(B);
    free(C);
}